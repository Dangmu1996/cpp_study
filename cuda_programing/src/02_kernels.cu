
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void VecAdd(float* A, float* B, float* C)
{
    // thread index로 더하기
    int i = threadIdx.x; // 이렇게 하면 thread index가 계산이 된다고 함.
    C[i] = A[i] + B[i];
}

/*    
1. 호스트 메모리에서 데이터 배열을 위한 공간을 할당한다.
2. 디바이스 메모리에 데이터 배열을 위한 공간을 할당한다.
3. 호스트에서 디바이스 메모리로 데이터를 복사합니다.
4. GPU 커널을 실행합니다.
5. 계산된 결과를 디바이스에서 호스트로 다시 복사합니다.
6. 디바이스 메모리 할당을 해제합니다.
*/

int main()
{
    float a[3] = {1, 1, 1};
    float b[3] = {1, 2, 3};
    float c[3];

    float *d_A, *d_B, *d_C;
    int size = 3 * sizeof(float);

    // 디바이스 메모리 할당
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 호스트에서 디바이스로 데이터 복사
    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);

    // 커널 실행
    VecAdd<<<1, 3>>>(d_A, d_B, d_C);

    // 디바이스에서 호스트로 결과 복사
    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost); //cudaMemcpy() 함수 자체가 블로킹 호출이기 때문에, GPU에서 커널 실행이 완료되기 전까지는 CPU 쓰레드가 대기 상태가 된다.

    cout << c[0] << ", "
         << c[1] << ", "
         << c[2] << ", "
         << endl;

    // 디바이스 메모리 해제
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}