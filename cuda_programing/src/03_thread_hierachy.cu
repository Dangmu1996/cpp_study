#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 4;

__global__ void MatAdd(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i * N + j;

    if (i < N && j < N)
        C[index] = A[index] + B[index];
}

int main()
{
    float a[N][N] = {
                    {1, 1, 1, 1},
                    {1, 1, 1, 1},
                    {1, 1, 1, 1},
                    {1, 1, 1, 1}};
    float b[N][N] = {
                    {1, 2, 3, 4},
                    {5, 6, 7, 8},
                    {9, 10, 11, 12},
                    {13, 14, 15, 16}};
    float c[N][N];

    float *d_A, *d_B, *d_C;
    int size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(c, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cout << c[i][j] << " ";
        }
        cout << endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
