#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 2;
const int M = 2;

// Kernel definition
__global__ void addArrays(float *input1, float *input2, float *output)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int idz = threadIdx.z + blockIdx.z * blockDim.z;

    int index = idx + idy * N + idz * N * M;

    if (idx < N && idy < M && idz < M)
    {
        output[index] = input1[index] + input2[index];
    }
}

int main()
{
    // 입력과 출력을 위한 메모리 할당
    float *input1, *input2, *output;
    hipMallocManaged(&input1, N * M * M * sizeof(float));
    hipMallocManaged(&input2, N * M * M * sizeof(float));
    hipMallocManaged(&output, N * M * M * sizeof(float));

    // 입력 데이터 초기화
    for (int i = 0; i < N * M * M; ++i)
    {
        input1[i] = i;
        input2[i] = 2 * i;
    }

    dim3 threadsPerBlock(2, 2, 1);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (M + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (M + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // 커널 실행
    addArrays<<<numBlocks, threadsPerBlock>>>(input1, input2, output);
    hipDeviceSynchronize(); // 커널 실행 완료 대기

    // 결과 출력
    cout << "덧셈 결과:" << endl;
    for (int i = 0; i < N * M * M; ++i)
    {
        cout << output[i] << " ";
    }
    cout << endl;

    // 메모리 해제
    hipFree(input1);
    hipFree(input2);
    hipFree(output);

    return 0;
}
