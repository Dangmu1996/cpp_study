
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

// 예를 kernel이라고 부름
__global__ void cuda_hello(){
    printf("hello world from gpu!\n");
}

int main(){
    cuda_hello<<<1,1>>>(); // <block, theard> cuda hello 실행
    hipDeviceSynchronize(); // 수는 CUDA 프로그램에서 모든 GPU 작업이 완료될 때까지 CPU 쓰레드의 실행을 차단(대기)하게 만드는 역할
    return 0;
}