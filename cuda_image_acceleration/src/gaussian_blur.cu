#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv4/opencv2/opencv.hpp>
#include "gaussian_blur.h"


__global__ void gaussianBlurKernel(const uchar3* inputImage, uchar3* outputImage, int width, int height) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f;
        int numPixels = 0;

        for (int offsetY = -1; offsetY <= 1; ++offsetY) {
            for (int offsetX = -1; offsetX <= 1; ++offsetX) {
                int neighborX = x + offsetX;
                int neighborY = y + offsetY;

                if (neighborX >= 0 && neighborX < width && neighborY >= 0 && neighborY < height) {
                    uchar3 neighborPixel = inputImage[neighborY * width + neighborX];
                    sumR += neighborPixel.x;
                    sumG += neighborPixel.y;
                    sumB += neighborPixel.z;
                    ++numPixels;
                }
            }
        }

        outputImage[y * width + x].x = sumR / numPixels;
        outputImage[y * width + x].y = sumG / numPixels;
        outputImage[y * width + x].z = sumB / numPixels;
    }
}

void gaussianBlurCUDA(const cv::Mat& inputImage, cv::Mat& outputImage) {
    // 이미지를 GPU로 복사
    uchar3* d_inputImage;
    uchar3* d_outputImage;
    size_t numPixels = inputImage.cols * inputImage.rows;
    hipMalloc(&d_inputImage, numPixels * sizeof(uchar3));
    hipMalloc(&d_outputImage, numPixels * sizeof(uchar3));
    hipMemcpy(d_inputImage, inputImage.ptr(), numPixels * sizeof(uchar3), hipMemcpyHostToDevice);

    // CUDA 커널 실행
    dim3 threadsPerBlock(32, 32);
    dim3 gridSize((inputImage.cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (inputImage.rows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    gaussianBlurKernel<<<gridSize, threadsPerBlock>>>(d_inputImage, d_outputImage, inputImage.cols, inputImage.rows);

    // 결과 이미지를 호스트로 복사
    outputImage.create(inputImage.size(), inputImage.type());
    hipMemcpy(outputImage.ptr(), d_outputImage, numPixels * sizeof(uchar3), hipMemcpyDeviceToHost);

    // 메모리 해제
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}
